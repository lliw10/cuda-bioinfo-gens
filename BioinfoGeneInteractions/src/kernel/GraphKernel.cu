#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <math.h>
#include "KernelUtils.h"
#include "HostUtils.h"
#include "GraphKernel.h"

__global__ void kernelLabelComponents( long long int *graph,
		unsigned long long int *components,
		int *hasChange,
		long long int sizeComponents,
		long long offset ) {

	unsigned long long int tid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long long int idx = tid;
	if( idx < sizeComponents ) {
		unsigned long long int ev1 = idx + offset;
		unsigned long long int ev2 = graph[idx];

		unsigned long long int cv1 = components[ev1];
		unsigned long long int cv2 = components[ev2];

		if( cv1 < cv2 ) {
			//			atomicMin( &components[e.v2], cv1 );
			components[ev2] = cv1;
			hasChange[0] = 1;
		} else if( cv1 > cv2 ) {
			//			atomicMin( &components[e.v1], cv2 );
			components[ev1] = cv2;
			hasChange[0] = 1;
		}
	}
}

__global__ void kernelInitializeConectedComponents( unsigned long long int *components,
		long long int sizeComponents,
		long long int offset ) {

	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	long long int idx = tid + offset;

	if( idx < sizeComponents ) {
		// comentar (somente para kernelSharedMemory)
		//		idx = (idx ^ (idx >> 1));
		components[idx] = idx;
	}
}

unsigned long long int * executeKernelLabelComponents( long long int *graph,
		unsigned long long int nVertices ) {

	printf( "....Start: executeKernelLabelComponents \n\n" );

	unsigned long long int *components = NULL;
	int *hasChangeHost = NULL;

	long long int *graphDev = NULL;
	unsigned long long int *componentsDev;
	int *hasChangeDev = NULL;

	long long int numThreadsPerBlock;
	long long int numBlocksOnGrid;

	long long int restThreadsToExecute;
	long long int numIterations;

	// Part 1 of 6: define kernel configuration
	// Number of threads per block

	// Part 3 of 6: allocate device memory
	long long int memComponentsSize = nVertices
			* (long long int) sizeof(long long int);
	int memHasChangeSize = sizeof(int);

	hasChangeHost = getPointerToMatrix( 1 );
	hipMalloc( (void **) &hasChangeDev, memHasChangeSize );
	if( checkCUDAError( "GraphKernel::hipMalloc. Aborting..." ) ) {
		return NULL;
	}

	//------- Allocate Zero Copy memory -------
	hipHostAlloc( (void **) &components, memComponentsSize,
			hipHostMallocMapped );
	if( !components || checkCUDAError(
			"GraphKernel::hipHostAlloc. Aborting..." ) ) {
		if( !components ) {
			printf(
					"GraphKernel::hipHostAlloc. Cannot allocate memory of size: %lld.\n",
					memComponentsSize );
		}
		return NULL;
	}

	hipHostGetDevicePointer( (void **) &componentsDev, (void *) components, 0 );
	if( !componentsDev || checkCUDAError(
			"GraphKernel::hipHostGetDevicePointer. Aborting..." ) ) {
		printf(
				"GraphKernel::hipHostGetDevicePointer. Cannot allocate memory of size: %lld.\n",
				memComponentsSize );
		return NULL;
	}

	//-------

	//------- Kernel Initialization Execution
	hipEvent_t start, stop;
	float time;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );

	calculateKernelLaunchConfiguration( (long long int) nVertices,
			(long long int *) &numThreadsPerBlock,
			(long long int *) &numBlocksOnGrid,
			(long long int *) &numIterations,
			(long long int *) &restThreadsToExecute );

	dim3 dimBlockInitVertices( numThreadsPerBlock );
	dim3 dimGridInitVertices( numBlocksOnGrid );

	long long int offset = 0;
	int numIterationsKernelInit = numIterations
			+ (restThreadsToExecute <= 0 ? 0 : 1);

	for (int i = 0; i < numIterationsKernelInit; i++) {
		kernelInitializeConectedComponents <<< dimGridInitVertices, dimBlockInitVertices>>> ( componentsDev, nVertices, offset );
		offset += dimBlockInitVertices.x * dimGridInitVertices.x;
	}

	hipDeviceSynchronize();
	checkCUDAError( "Error: kernelInitializeConectedComponents" );

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	float timeKernelInitialize = (time / 1000);

	int steps = 0;

	hipEvent_t startTotalTime, stopTotalTime;
	hipEventCreate( &startTotalTime );
	hipEventCreate( &stopTotalTime );
	hipEventRecord( startTotalTime, 0 );
	//-------

	//------- Kernel label components execution

	calculateKernelLaunchConfiguration( (long long int) nVertices,
			&numThreadsPerBlock, &numBlocksOnGrid, &numIterations,
			&restThreadsToExecute );

	dim3 dimBlockLabelComp( numThreadsPerBlock );
	dim3 dimGridLabelComp( numBlocksOnGrid );

	printf( "Number of blocks used: %lld \n", numBlocksOnGrid );
	printf( "Number of threads used: %lld \n", numThreadsPerBlock );
	printf( "Internal iterations: %lld \n", numIterations );

	long long int previousMemGraphSize = -1;

	do {

		if( steps % 1000 == 0 ) {
			hipEventCreate( &start );
			hipEventCreate( &stop );
			hipEventRecord( start, 0 );
		}

		steps++;
		hasChangeHost[0] = 0;

		hipMemcpy( hasChangeDev, hasChangeHost, memHasChangeSize,
				hipMemcpyHostToDevice );

		offset = 0;
		for (int i = 0; i < numIterationsKernelInit; i++) {

			// Part 4 of 6: host to device copy
			int graphDevSize;
			if( i == (numIterationsKernelInit - 1) && restThreadsToExecute > 0 ) {
				graphDevSize = restThreadsToExecute;

				calculateKernelLaunchConfiguration( graphDevSize,
						&numThreadsPerBlock, &numBlocksOnGrid, &numIterations,
						&restThreadsToExecute );

				dim3 dimBlock2( numThreadsPerBlock );
				dim3 dimGrid2( numBlocksOnGrid );

				dimBlockLabelComp = dimBlock2;
				dimGridLabelComp = dimGrid2;

			} else {
				graphDevSize = (dimGridLabelComp.x * dimBlockLabelComp.x);
			}

			long long int memGraphSize = graphDevSize
					* (long long int) sizeof(long long int);

			long long int *partialGraph = (graph + offset);

			if( previousMemGraphSize != memGraphSize ) {
				if( graphDev != NULL ) {
					hipFree( graphDev );
					checkCUDAError( "edgesDev [partial] Free" );
				}
				printf( "GraphPartial size [MEM: %f MB] \n",
						((double) (memGraphSize) / (1024.0 * 1024.0)) );

				hipMalloc( (void **) &graphDev, memGraphSize );
				previousMemGraphSize = memGraphSize;
			}
			hipMemcpy( graphDev, partialGraph, memGraphSize,
					hipMemcpyHostToDevice );
			checkCUDAError( "edgesDev Memory Allocation" );

			kernelLabelComponents <<< dimGridLabelComp, dimBlockLabelComp>>>(graphDev, componentsDev, hasChangeDev, graphDevSize, offset);
			offset += dimGridLabelComp.x * dimBlockLabelComp.x;

			hipDeviceSynchronize();
			checkCUDAError( "Kernel execution" );

		}

		hipMemcpy( hasChangeHost, hasChangeDev, memHasChangeSize,
				hipMemcpyDeviceToHost );

		checkCUDAError( "Memory copy" );

		if( steps % 1000 == 0 ) {
			hipEventRecord( stop, 0 );
			hipEventSynchronize( stop );
			hipEventElapsedTime( &time, start, stop );
			hipEventDestroy( start );
			hipEventDestroy( stop );
			//			printf( "step: %d - kernelLabelComponents time %f s \n", steps,
			//					(time / 1000) );
		}

	} while( hasChangeHost[0] == 1 );

	offset += dimGridLabelComp.x * dimBlockLabelComp.x;

	hipDeviceSynchronize();
	hipMemcpy( components, componentsDev, memComponentsSize,
			hipMemcpyDeviceToHost );

	checkCUDAError( "Memory copy verticesComponentOut " );

	// Calculate total time for algorithm execution
	hipEventRecord( stopTotalTime, 0 );
	hipEventSynchronize( stopTotalTime );
	hipEventElapsedTime( &time, startTotalTime, stopTotalTime );
	hipEventDestroy( startTotalTime );
	hipEventDestroy( stopTotalTime );
	float timeKernelLabelComponents = (time / 1000);

	printf( "kernelInitializeConectedComponents time (s): %f \n",
			timeKernelInitialize );
	printf( "kernelLabelComponents time (s): %f \n", timeKernelLabelComponents );
	printf( "Number of steps: %d \n", steps );

	// free device memory
	if( graphDev != NULL ) {
		hipFree( graphDev );
		checkCUDAError( "edgesDev [end] Free" );
	}
	hipFree( hasChangeDev );
	checkCUDAError( "hasChangeDev Free" );

	printf( "\n\n....End: executeKernelLabelComponents\n\n" );

	return components;
}
